#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <cstdlib>

#include "Primes.CUDA.LIB.h"
#include "Primes.CUDA.cuh"

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

int main() { return 0; }

uint8_t* parallel_is_prime(uint64_t start, int32_t threadsPerBlock, int32_t blocksPerGrid, int32_t perThreadMult)
{
	size_t size = threadsPerBlock * blocksPerGrid * perThreadMult * sizeof(uint8_t);

	//allocate buffer in host memory
	uint8_t* host_buffer = (uint8_t*)malloc(size);

	//alocate buffer in device memory
	uint8_t* device_buffer;
	hipMalloc(&device_buffer, size);


	//call kernel
	//DO NOT MULT PERTHREAD BY 8 OR YOU WILL FUCK IT UP (done in kernel)
	check_primes_kernel <<<blocksPerGrid, threadsPerBlock>>>(start, perThreadMult, device_buffer);


	//get data
	hipMemcpy(host_buffer, device_buffer, size, hipMemcpyDeviceToHost);
	//free memory (else mem leak ;))
	hipFree(device_buffer);


	//free host memory?
	return host_buffer;
}
